#include <fstream>
#include <sstream>
#include <vector>
#include <string>

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>

#include "matrix_mult_gpu.cu"


// Helper function for reading input file
void readMatrixFile(const char* file, std::vector<float> &matrix, int &row, int &col){
    // TODO
}

int MatrixMult(int argc, int **argv, int block_size, const dim3 &dimsA,
                   const dim3 &dimsB){
    unsigned int sizeA = dimsA.x * dimsB.y;
    unsigned int mem_sizeA = sizeA * sizeof(float);
    checkCudaErrors(hipHostMalloc(&h_A, mem_size_A));
    float *h_A;


}